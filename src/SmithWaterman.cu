#include "hip/hip_runtime.h"
#include "SmithWaterman.ch"

#include <cstdio>


#define CUERR do{ hipError_t err;    .\
if ((err = hipGetLastError()) != hipSuccess) {    \
            int device;\
        hipGetDevice(&device);\
    printf("CUDA error on GPU %d: %s : %s, line %d\n", device, hipGetErrorString(err), __FILE__, __LINE__); }}while(0);

__global__ void searchCUDA() {
    printf("%d", threadIdx.x);
}

SmithWaterman::SmithWaterman(Params &params) {

}

SmithWaterman::~SmithWaterman() {

}

void SmithWaterman::search(dim3 grid, dim3 block) {
    searchCUDA<<<grid, block>>>();
}


