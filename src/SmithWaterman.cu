#include "hip/hip_runtime.h"
#include "SmithWaterman.h"

#include <cstdio>


#define CUERR do{ hipError_t err;    .\
if ((err = hipGetLastError()) != hipSuccess) {    \
            int device;\
        hipGetDevice(&device);\
    printf("CUDA error on GPU %d: %s : %s, line %d\n", device, hipGetErrorString(err), __FILE__, __LINE__); }}while(0);

__global__ void searchCUDA() {
    printf("%d", threadIdx.x);
    

}

SmithWaterman::SmithWaterman() {

}

SmithWaterman::~SmithWaterman() {

}

void SmithWaterman::search(Params &params) {
    

    dim3 grid( 3 );
    dim3 block( 3, 32 );
    searchCUDA<<<grid, block>>>();
    uint *g_H;
}


