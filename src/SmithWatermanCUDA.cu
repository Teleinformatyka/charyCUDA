

#include <hip/hip_runtime.h>
#include <cstdio>


#define CUERR do{ hipError_t err;    .\
if ((err = hipGetLastError()) != hipSuccess) {    \
            int device;\
        hipGetDevice(&device);\
    printf("CUDA error on GPU %d: %s : %s, line %d\n", device, hipGetErrorString(err), __FILE__, __LINE__); }}while(0);

__global__ void runCUDA() {
    printf("%d", threadIdx.x);
    

}
void searchCUDA() {

    runCUDA<<<20, 30>>>();
    

}

